
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <cstdint>

__global__ void gpuProcess(int n, double *arr){
    double localMax = -1;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){
        if (arr[i] > localMax) localMax = arr[i];
    }

    arr[index] = localMax;
}

int main(void){
    int N = 50000000;
    double *arr;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&arr, N*sizeof(double));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        double r = rand()/1000000.0;
        arr[i] = r;
    }

    int numThreads = N;
    int threadsPerBlock = 256;

    do {
        numThreads = N/16;
        if (numThreads == 0) numThreads = 1;
        if (numThreads < threadsPerBlock) threadsPerBlock = numThreads;
        int numBlocks = (numThreads + threadsPerBlock - 1)/threadsPerBlock;
        gpuProcess<<<numBlocks, threadsPerBlock>>>(N, arr);

        std::cout << "Launching " << numThreads << " threads: " << numBlocks << " blocks and " << threadsPerBlock << " threads/block" << std::endl;

        hipDeviceSynchronize();
        N = numBlocks * threadsPerBlock;
    } while(numThreads > 1);
    

    std::cout << "MAX: " << arr[0] << std::endl;

    // Free memory
    hipFree(arr);
    return 0;
}