
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <cstdint>
#include <time.h>
#include <cstdio>

void cpuProcess(int n, double *arr){
    double localMax = -1;
    
    for (int i = 0; i < n; i ++){
        if (arr[i] > localMax) localMax = arr[i];
    }
    
    arr[0] = localMax;
}

__global__ void gpuProcess(int n, double *arr){
    double localMax = -1;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){
        if (arr[i] > localMax) localMax = arr[i];
    }

    arr[index] = localMax;
}

int main(void){
    clock_t start, diff;
    int N = 200000000;

    double *h_arr = new double[N];
    for (int i = 0; i < N; i++) {
        double r = rand()/1000000.0;
        h_arr[i] = r;
    }

    start = clock();
    cpuProcess(N, h_arr);
    diff = (clock() - start) * 1000 / CLOCKS_PER_SEC;
    std::cout << "CPU MAX: " << h_arr[0] << std::endl;
    printf("Time taken for cpu: %d milliseconds\n\n", diff);

    start = clock();
    double *d_arr;
    hipMalloc(&d_arr, sizeof(double)*N);
    hipMemcpy(d_arr, h_arr, N*sizeof(double), hipMemcpyHostToDevice);
    diff = (clock() - start) * 1000 / CLOCKS_PER_SEC;
    printf("Time taken to copy arr to gpu: %d milliseconds\n", diff);
  
    int numThreads = N;
    int threadsPerBlock = 256;

    start = clock();
    do {
        numThreads = N/16;
        if (numThreads == 0) numThreads = 1;
        if (numThreads < threadsPerBlock) threadsPerBlock = numThreads;
        int numBlocks = (numThreads + threadsPerBlock - 1)/threadsPerBlock;
        gpuProcess<<<numBlocks, threadsPerBlock>>>(N, d_arr);
        std::cout << "Launching " << numThreads << " threads: " << numBlocks << " blocks and " << threadsPerBlock << " threads/block" << std::endl;
        hipDeviceSynchronize();

        N = numBlocks * threadsPerBlock;
    } while(numThreads > 1);
    
    hipMemcpy(h_arr, d_arr, 1*sizeof(double), hipMemcpyDeviceToHost);
    std::cout << "GPU MAX: " << h_arr[0] << std::endl;
    diff = (clock() - start) * 1000 / CLOCKS_PER_SEC;
    printf("Time taken for gpu: %d milliseconds\n", diff);

    // Free memory
    hipFree(d_arr);
    return 0;
}