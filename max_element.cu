
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <cstdint>


__global__ void gpuProcess(int n, double *arr){
    double localMax = -1;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){
        if (arr[i] > localMax) localMax = arr[i];
    }

    arr[index] = localMax;
}

int main(void){

    int N = 50000000;
    double *arr;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&arr, N*sizeof(double));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        double r = rand()/1000000.0;
        arr[i] = r;
    }

    // Run kernel on 1M elements on the GPU
    gpuProcess<<<1000, 512>>>(N, arr);
    hipDeviceSynchronize();

    std::cout << "MAX: " << arr[0] << std::endl;

    // Free memory
    hipFree(arr);
    return 0;
}