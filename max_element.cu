
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <cstdint>

__global__ void gpuProcess(int n, double *arr){
    double localMax = -1;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){
        if (arr[i] > localMax) localMax = arr[i];
    }

    arr[index] = localMax;
}

int main(void){
    int N = 50000000;

    double *h_arr = new double[N];
    for (int i = 0; i < N; i++) {
        double r = rand()/1000000.0;
        h_arr[i] = r;
    }

    double *d_arr;
    hipMalloc(&d_arr, sizeof(double)*N);
    hipMemcpy(d_arr, h_arr, N*sizeof(double), hipMemcpyHostToDevice);
  

    int numThreads = N;
    int threadsPerBlock = 256;

    do {
        numThreads = N/16;
        if (numThreads == 0) numThreads = 1;
        if (numThreads < threadsPerBlock) threadsPerBlock = numThreads;
        int numBlocks = (numThreads + threadsPerBlock - 1)/threadsPerBlock;
        gpuProcess<<<numBlocks, threadsPerBlock>>>(N, d_arr);
        std::cout << "Launching " << numThreads << " threads: " << numBlocks << " blocks and " << threadsPerBlock << " threads/block" << std::endl;
        hipDeviceSynchronize();

        N = numBlocks * threadsPerBlock;
    } while(numThreads > 1);
    
    hipMemcpy(h_arr, d_arr, N*sizeof(double), hipMemcpyDeviceToHost);
    std::cout << "MAX: " << h_arr[0] << std::endl;

    // Free memory
    hipFree(d_arr);
    return 0;
}