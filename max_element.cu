#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

__device__ double gMax, gMin;

__device__ void AtomicMax(double * const address, const double value)
{
	if (* address >= value)
	{
		return;
	}

	uint64 * const address_as_i = (uint64 *)address;
    uint64 old = * address_as_i, assumed;

	do 
	{
        assumed = old;
		if (__longlong_as_double(assumed) >= value)
		{
			break;
		}
		
        old = atomicCAS(address_as_i, assumed, __double_as_longlong(value));
    } while (assumed != old);
}

__global__ void initVars(){
    gMax = -1;
    gMin = 1001;
}

__global__ void gpuMax(int n, float *arr){
    double localMax = -1, localMin = 1001;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){
        if (arr[i] > localMax) localMax = arr[i];
        if (arr[i] < localMin) localMin = arr[i];
    }
}

int main(void){

    int N = 5000000;
    double *arr;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&arr, N*sizeof(double));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        double r = (rand()/(double)RAND_MAX) * 1000.0;
        arr[i] = r;
    }

    // Run kernel on 1M elements on the GPU
    gpuMax<<<1, 1>>>(N, arr);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);
  
    return 0;
}