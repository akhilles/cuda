#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <cstdint>


__global__ void gpuProcess(int n, double *arr){
    double localMax = -1;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){
        if (arr[i] > localMax) localMax = arr[i];
    }

    arr[index] = localMax;
}

int main(void){

    int N = 5000000;
    double *arr;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&arr, N*sizeof(double));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        double r = (rand()/(double)RAND_MAX) * 1000.0;
        arr[i] = r;
    }

    initVars<<<1, 1>>>();
    hipDeviceSynchronize();
    // Run kernel on 1M elements on the GPU
    gpuProcess<<<1, 1>>>(N, arr);
    hipDeviceSynchronize();

    std::cout << "MAX: " << arr[0] << std::endl;

    // Free memory
    hipFree(arr);
    return 0;
}