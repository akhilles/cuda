#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

__device__ double d_max, d_min;

__device__ void AtomicMax(double * const address, const double value){
	if (* address >= value) return
	uint64 * const address_as_i = (uint64 *)address;
    uint64 old = * address_as_i, assumed;
	do {
        assumed = old;
		if (__longlong_as_double(assumed) >= value) break;
        old = atomicCAS(address_as_i, assumed, __double_as_longlong(value));
    } while (assumed != old);
}

__device__ void AtomicMin(double * const address, const double value){
	if (* address <= value) return
	uint64 * const address_as_i = (uint64 *)address;
    uint64 old = * address_as_i, assumed;
	do {
        assumed = old;
		if (__longlong_as_double(assumed) <= value) break;
        old = atomicCAS(address_as_i, assumed, __double_as_longlong(value));
    } while (assumed != old);
}

__global__ void initVars(){
    d_max = -1;
    d_min = 1001;
}

__global__ void gpuProcess(int n, float *arr){
    double localMax = -1, localMin = 1001;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){
        if (arr[i] > localMax) localMax = arr[i];
        if (arr[i] < localMin) localMin = arr[i];
    }

    AtomicMax(&d_max, localMax)
    AtomicMin(&d_min, localMin)
}

int main(void){

    int N = 5000000;
    double *arr;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&arr, N*sizeof(double));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        double r = (rand()/(double)RAND_MAX) * 1000.0;
        arr[i] = r;
    }

    initVars<<<1, 1>>>();
    hipDeviceSynchronize();
    // Run kernel on 1M elements on the GPU
    gpuProcess<<<1, 1>>>(N, arr);
    hipDeviceSynchronize();

    typeof(d_max) h_max;
    hipMemcpyFromSymbol(&h_max, HIP_SYMBOL("d_max"), sizeof(h_max), 0, hipMemcpyDeviceToHost);
    std::cout << "MAX: " << h_max << std::endl;

    // Free memory
    hipFree(arr);
    return 0;
}