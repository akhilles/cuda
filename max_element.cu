
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <cstdint>

#define THREADS_PER_BLOCK 256

__global__ void gpuProcess(int n, double *arr){
    double localMax = -1;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){
        if (arr[i] > localMax) localMax = arr[i];
    }

    arr[index] = localMax;
}

int main(void){

    int N = 50000000;
    double *arr;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&arr, N*sizeof(double));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        double r = rand()/1000000.0;
        arr[i] = r;
    }

    int numThreads = N;

    do {
        numThreads /= 10;
        if (numThreads == 0) numThreads = 1;
        std::cout << "Launching " << numThreads << " threads" << std::endl;
        gpuProcess<<<(numThreads + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(N, arr);
        hipDeviceSynchronize();
        N = numThreads;
    } while(numThreads > 1);
    

    std::cout << "MAX: " << arr[0] << std::endl;

    // Free memory
    hipFree(arr);
    return 0;
}